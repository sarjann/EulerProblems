
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <algorithm>


#define N 1000
#define N_MAX 1000000

__global__ void add_vectors(int *inputs, long *outputs) {
    int i = inputs[threadIdx.x];
    int max = i + N;

    while (i < max){
        int length = 1;
        long start = i;

        if (start == 0){
            start = 1;
        }

        while (start != 1){
            if (start % 2 == 0){
                start = start / 2;
            }
            else{
                start = 3 * start + 1;
            }
            length += 1;
        }
        outputs[i] = length;
        i += 1;
    }
}

int main(void) {
    int *inputs;
    long *outputs;
    int *d_inputs;
    long *d_outputs;

    int size = N * sizeof(int);
    int size_max = N_MAX * sizeof(long);

    inputs = (int *)malloc(size);
    outputs = (long *)malloc(size_max);

    for (int i = 0; i < N; ++i){
        inputs[i] = N*i;
    }


    hipMalloc(&d_inputs, size);
    hipMalloc(&d_outputs, size_max);

    hipMemcpy(d_inputs, inputs, size, hipMemcpyHostToDevice);

    add_vectors<<<1, N>>>(d_inputs, d_outputs);

    hipMemcpy(outputs, d_outputs, size_max, hipMemcpyDeviceToHost);

    hipFree(d_inputs);
    hipFree(d_outputs);

//    for (int i = 0; i < 10000; ++i){
//        std::cout << outputs[i] << std::endl;
//    }

    std::cout << "------------" << std::endl;

    long max = 0;
    int ind_max = 0;
    for (int i = 0; i < N_MAX + 1; i++){
        if (outputs[i] > max) {
            max = outputs[i];
            ind_max = i;
        }
    }

    std::cout << max << std::endl;
    std::cout << ind_max << std::endl;

    free(inputs);
    free(outputs);

    return 0;
}