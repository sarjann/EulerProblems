
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <algorithm>


#define N 1000

__global__ void add_vectors(int *inputs, int *outputs) {
    int num = inputs[threadIdx.x];
    int count = 0;
    for (int i = 0; i <= num; i++){
        for (int j = 0; j <= num - i; j++){
            int k = num - i - j;
                if (i*i + j*j - k*k == 0){
                    count += 1;
                }

        }
    }
    outputs[threadIdx.x] = count;
}

int main(void) {
    int *inputs;
    int *outputs;
    int *d_inputs;
    int *d_outputs;

    int size = N * sizeof(int);

    inputs = (int *)malloc(size);
    outputs = (int *)malloc(size);

    for (int i = 0; i < N; ++i){
        inputs[i] = i + 1;
    }


    hipMalloc(&d_inputs, size);
    hipMalloc(&d_outputs, size);

    hipMemcpy(d_inputs, inputs, size, hipMemcpyHostToDevice);

    add_vectors<<<1, N>>>(d_inputs, d_outputs);

    hipMemcpy(outputs, d_outputs, size, hipMemcpyDeviceToHost);

    hipFree(d_inputs);
    hipFree(d_outputs);

    for (int i = 0; i < 1000; ++i){
        std::cout << i + 1 << ": " << outputs[i] << std::endl;
    }

    std::cout << "------------" << std::endl;

    int max = 0;
    int num_max = 0;
    for (int i = 0; i < N; i++){
        if (outputs[i] > max) {
            max = outputs[i];
            num_max = i + 1;
        }
    }

    std::cout << max << std::endl;
    std::cout << num_max << std::endl;

    free(inputs);
    free(outputs);

    return 0;
}